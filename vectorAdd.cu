#include "hip/hip_runtime.h"

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include "lib/hip/hip_runtime_api.h"
#include "hip/hip_complex.h"
#include "hip/hip_fp16.h"
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

//select variable type: 1=real, 2=complex32, 3=half2, 4=2x half
#define TYPE 4

#if TYPE==1
    #define VARTYPE float
#elif TYPE==2
    #define VARTYPE hipComplex
#elif TYPE==3
    #define VARTYPE __half2
#else 
    #define VARTYPE cmplx16 
#endif


struct cmplx16 {
    __half x;
    __half y;
};


template <typename T>
__global__ void vectorAdd(const T *A, const T *B, T *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

template <>
__global__ void vectorAdd<hipComplex>(const hipComplex* A, const hipComplex* B, hipComplex* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i].x = A[i].x + B[i].x;
        C[i].y = A[i].y + B[i].y;
    }
}

template<>
__global__ void vectorAdd<cmplx16>(const cmplx16* A, const cmplx16* B, cmplx16* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
#if __CUDA_ARCH__ >= 530
        C[i].x = __hadd(A[i].x, B[i].x);
        C[i].y = __hadd(A[i].y, B[i].y);
#else
        C[i].x = __float2half(__half2float(A[i].x) + __half2float(B[i].x));
        C[i].y = __float2half(__half2float(A[i].y) + __half2float(B[i].y));
#endif
    }
}

template<>
__global__ void vectorAdd<__half2>(const __half2* A, const __half2* B, __half2* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
#if __CUDA_ARCH__ >= 530
        C[i] = __hadd2(A[i], B[i]);
#else
        C[i] = __floats2half2_rn(__half22float2(A[i]).x + __half22float2(B[i]).x,
            __half22float2(A[i]).y + __half22float2(B[i]).y);
#endif
    }
}


/**
 * Host main routine
 */
int
main(void)
{
	const int n_trials = 10000; // Number of montecarlo simulations to perform to get a better sense of the average stats. 

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(VARTYPE);
    printf("[Vector addition of %d elements]\n", numElements);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the host input vector A
    VARTYPE *h_A = (VARTYPE *)malloc(size);

    // Allocate the host input vector B
    VARTYPE *h_B = (VARTYPE *)malloc(size);

    // Allocate the host output vector C
    VARTYPE *h_C = (VARTYPE *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    // Initialize the host input vectors
    #if TYPE==1
    
        for (int i = 0; i < numElements; ++i)
        {
            h_A[i] = rand() / (float)RAND_MAX;
            h_B[i] = rand() / (float)RAND_MAX;
        }
    #elif TYPE==2 || TYPE==4
        for (int i = 0; i < numElements; ++i)
        {
            h_A[i].x = rand() / (float)RAND_MAX;
            h_A[i].y = rand() / (float)RAND_MAX;
            h_B[i].x = rand() / (float)RAND_MAX;
            h_B[i].y = rand() / (float)RAND_MAX;
        }
    #elif TYPE==3
        float2 temp_A_float2;
        float2 temp_B_float2;   
        for (int i = 0; i < numElements; ++i)
        {
            temp_A_float2.x = rand() / (float)RAND_MAX;
            temp_A_float2.y = rand() / (float)RAND_MAX;
            h_A[i] = __float22half2_rn(temp_A_float2);
            temp_B_float2.x = rand() / (float)RAND_MAX;
            temp_B_float2.y = rand() / (float)RAND_MAX;
            h_B[i] = __float22half2_rn(temp_B_float2);

        }
    #endif


    // Allocate the device input vector A
    VARTYPE *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    VARTYPE *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    VARTYPE *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Setup CUDA grid.
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	//For total elapsed time.
	float total_time = 0;

	for (int i = 0; i < n_trials; i++){
		// Copy the host input vectors A and B in host memory to the device input vectors in
		// device memory
		//printf("Copy input data from the host memory to the CUDA device\n");
		err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		hipEventRecord(start);
		err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		
		vectorAdd<VARTYPE><<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Copy the device result vector in device memory to the host result vector
		// in host memory.
		//printf("Copy output data from the CUDA device to the host memory\n");
		err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		hipEventRecord(stop); // Notice here your copy is only around the CUDA kernel execution. It is not around the memcopy.
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		total_time = milliseconds + total_time;
	}

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    //for (int i = 0; i < numElements; ++i)
    //{
    //    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    //    {
    //        fprintf(stderr, "Result verification failed at element %d!\n", i);
    //        exit(EXIT_FAILURE);
    //    }
    //}

    int idx1= 6;

    #if TYPE==1
        printf("Sample output on index %d: %f+%f=%f\n", idx1, h_A[idx1], h_B[idx1], h_C[idx1]);
    #elif TYPE==2
        printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, h_A[idx1].x, h_A[idx1].y, h_B[idx1].x, h_B[idx1].y, h_C[idx1].x, h_C[idx1].y);
    #elif TYPE==3
        float2 sample_val_A=__half22float2(h_A[idx1]);
        float2 sample_val_B=__half22float2(h_B[idx1]);
        float2 sample_val_C= __half22float2(h_C[idx1]);
    
        printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, sample_val_A.x, sample_val_A.y, sample_val_B.x, sample_val_B.y,
            sample_val_C.x, sample_val_C.y);
    #else
        printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, __half2float(h_A[idx1].x), __half2float(h_A[idx1].y),
            __half2float(h_B[idx1].x), __half2float(h_B[idx1].y), __half2float(h_C[idx1].x), __half2float(h_C[idx1].y));
    #endif


    printf("Total elapsed time including kernel execution and mem transfers from device to host: %f ms\n", total_time);
	auto average_time = total_time / n_trials;
	printf("Average time per trial: %f ms\n", average_time);

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

