#include "hip/hip_runtime.h"

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include "lib/hip/hip_runtime_api.h"
#include "hip/hip_complex.h"
#include "hip/hip_fp16.h"
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

//select variable type: 1=real, 2=complex32, 3=half2, 4=2x half
#define TYPE 1
#define COMPCAP 610  //gtx 9xx is 510, gtx 10xx is 610 (need to input manually since __CUDA_ARCH__ is undef in host code)

#if TYPE==1
    #define VARTYPE float
#elif TYPE==2
    #define VARTYPE hipComplex
#elif TYPE==3
    #define VARTYPE __half2
#else 
    #define VARTYPE cmplx16 
#endif


struct cmplx16 {
    __half x;
    __half y;
};

__host__ __device__ inline cmplx16 operator+(const cmplx16& a, const cmplx16& b) {
    #if __CUDA_ARCH__ >= 530
        const auto x = __hadd(a.x, b.x);
        const auto y = __hadd(a.y, b.y);
    #else
        const auto x = __float2half(__half2float(a.x) + __half2float(b.x));
        const auto y = __float2half(__half2float(a.y) + __half2float(b.y));
    #endif
    const  cmplx16 answer = {x, y};
    return answer;
}

__host__ __device__ inline hipComplex operator+(const hipComplex& a, const hipComplex& b) {
    const auto x = a.x + b.x;
    const auto y = a.y + b.y;
    const  hipComplex answer = {x, y};
    return answer;
}

#if COMPCAP>=530
#else 
__host__ __device__ inline __half2 operator+(const __half2& a, const __half2& b) {
        const __half2 answer = __floats2half2_rn(__half22float2(a).x + __half22float2(b).x,
            __half22float2(a).y + __half22float2(b).y);
        return answer;
}
#endif

template <typename T>
__global__ void vectorAdd(const T *A, const T *B, T *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}



/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(VARTYPE);
    printf("[Vector addition of %d elements]\n", numElements);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the host input vector A
    VARTYPE *h_A = (VARTYPE *)malloc(size);

    // Allocate the host input vector B
    VARTYPE *h_B = (VARTYPE *)malloc(size);

    // Allocate the host output vector C
    VARTYPE *h_C = (VARTYPE *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    // Initialize the host input vectors
    #if TYPE==1
    
        for (int i = 0; i < numElements; ++i)
        {
            h_A[i] = rand() / (float)RAND_MAX;
            h_B[i] = rand() / (float)RAND_MAX;
        }
    #elif TYPE==2 || TYPE==4
        for (int i = 0; i < numElements; ++i)
        {
            h_A[i].x = rand() / (float)RAND_MAX;
            h_A[i].y = rand() / (float)RAND_MAX;
            h_B[i].x = rand() / (float)RAND_MAX;
            h_B[i].y = rand() / (float)RAND_MAX;
        }
    #elif TYPE==3
        float2 temp_A_float2;
        float2 temp_B_float2;   
        for (int i = 0; i < numElements; ++i)
        {
            temp_A_float2.x = rand() / (float)RAND_MAX;
            temp_A_float2.y = rand() / (float)RAND_MAX;
            h_A[i] = __float22half2_rn(temp_A_float2);
            temp_B_float2.x = rand() / (float)RAND_MAX;
            temp_B_float2.y = rand() / (float)RAND_MAX;
            h_B[i] = __float22half2_rn(temp_B_float2);

        }
    #endif


    // Allocate the device input vector A
    VARTYPE *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    VARTYPE *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    VARTYPE *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    hipEventRecord(start);
    vectorAdd<VARTYPE><<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipEventRecord(stop);
    err = hipGetLastError();



    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    //for (int i = 0; i < numElements; ++i)
    //{
    //    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    //    {
    //        fprintf(stderr, "Result verification failed at element %d!\n", i);
    //        exit(EXIT_FAILURE);
    //    }
    //}

    int idx1= 6;

    #if TYPE==1
        printf("Sample output on index %d: %f+%f=%f\n", idx1, h_A[idx1], h_B[idx1], h_C[idx1]);
    #elif TYPE==2
        printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, h_A[idx1].x, h_A[idx1].y, h_B[idx1].x, h_B[idx1].y, h_C[idx1].x, h_C[idx1].y);
    #elif TYPE==3
        float2 sample_val_A=__half22float2(h_A[idx1]);
        float2 sample_val_B=__half22float2(h_B[idx1]);
        float2 sample_val_C= __half22float2(h_C[idx1]);
    
        printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, sample_val_A.x, sample_val_A.y, sample_val_B.x, sample_val_B.y,
            sample_val_C.x, sample_val_C.y);
    #else
        printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, __half2float(h_A[idx1].x), __half2float(h_A[idx1].y),
            __half2float(h_B[idx1].x), __half2float(h_B[idx1].y), __half2float(h_C[idx1].x), __half2float(h_C[idx1].y));
    #endif


    printf("Kernel time: %f ms\n", milliseconds);

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

